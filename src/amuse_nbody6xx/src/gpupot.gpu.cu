#include "hip/hip_runtime.h"
//#include <iostream>
#include <cstdio>
// #include <cutil.h>
#ifdef WITH_CUDA5
#  include <hip/hip_runtime_api.h>
#  define CUDA_SAFE_CALL checkCudaErrors
#else
#  include <cutil.h>
#endif
#include "cuda_pointer.h"
#define NTHREAD 128

#define PROFILE
#ifdef PROFILE
#include <sys/time.h>
static double get_wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.e-6 * tv.tv_usec;
}
#else
static double get_wtime(){
	return 0.0;
}
#endif

static float2 float2_split(double x){
	const int shift = 20;
	float2 ret;
	x *= (1<<shift);
	double xi = (int)x;
	double xf = x - xi;
	ret.x = xi * (1./(1<<shift));
	ret.y = xf * (1./(1<<shift));
	return ret;
}
__device__ float2 float2_accum(float2 acc, float x){
	float tmp = acc.x + x;
	acc.y -= (tmp - acc.x) - x;
	acc.x = tmp;
	return acc;
}

__device__ float2 float2_regularize(float2 acc){
	float tmp = acc.x + acc.y;
	acc.y = acc.y -(tmp - acc.x);
	acc.x = tmp;
	return acc;
}

struct Particle{
	float2 pos[3];
	float mass;
	float pad;

	Particle(double x[3], double m){
		pos[0] = float2_split(x[0]);
		pos[1] = float2_split(x[1]);
		pos[2] = float2_split(x[2]);
		mass = (float)m;
	}
	Particle(int){
		pos[0].x = pos[0].y = pos[1].x = pos[1].y = pos[2].x = pos[2].y = mass = pad = 0.f;
	}
	__device__ Particle() {}
};

__global__ void pot_kernel(int n, int istart, Particle *ptcl, float2 *phi){
	__shared__ Particle jpbuf[NTHREAD];
	int i = NTHREAD * blockIdx.x + threadIdx.x;
	Particle ip = ptcl[i+istart-1];
	float2 phii = make_float2(0.f, 0.f);
	for(int j=0; j<n; j+= NTHREAD){
		__syncthreads();
		jpbuf[threadIdx.x] = ptcl[j + threadIdx.x];
		__syncthreads();
#pragma unroll 4
		for(int jj=0; jj<NTHREAD; jj++){
			// if(j+jj == i) continue;
			Particle &jp = jpbuf[jj];
			float dx = (jp.pos[0].x - ip.pos[0].x) + (jp.pos[0].y - ip.pos[0].y);
			float dy = (jp.pos[1].x - ip.pos[1].x) + (jp.pos[1].y - ip.pos[1].y);
			float dz = (jp.pos[2].x - ip.pos[2].x) + (jp.pos[2].y - ip.pos[2].y);
			float r2 = dx*dx + dy*dy + dz*dz;
			// if(r2==0.f) continue;
			float pij = jp.mass * rsqrtf(r2);
			// phii = float2_accum(phii, pij);
			if(r2 > 0.f) phii = float2_accum(phii, pij);
		}
		phii = float2_regularize(phii);
	}
	phi[i] = phii;
}

extern "C"  void gpunb_devinit_(int *irank);

void gpupot(
        int *rank,
        int istart,
        int ni,
		int n,
		double m[],
		double x[][3],
		double pot[]){
	gpunb_devinit_(rank);

	double t0 = get_wtime();
	cudaPointer <float2> phi;
	cudaPointer <Particle> ptcl;
	int ng = NTHREAD * (ni/NTHREAD + (ni%NTHREAD ? 1 : 0));
    int ntg = NTHREAD * (n/NTHREAD + (n%NTHREAD ? 1 : 0));

	phi.allocate(ng);
	ptcl.allocate(ntg);

    //    std::cout << n << " " << ng << " "<< ntg << std::endl;
	for(int i=0; i<n; i++){
		// ptcl_h[i] = Particle(x[i], m[i]);
		ptcl[i] = Particle(x[i], m[i]);
	}
	for(int i=n; i<ntg; i++){
		// ptcl_h[i] = Particle(0);
		ptcl[i] = Particle(0);
	}

	// hipMemcpy(ptcl_d, ptcl_h, ng * sizeof(Particle), hipMemcpyHostToDevice);
	ptcl.htod(ntg);
	
	dim3 grid(ng/NTHREAD, 1, 1);
	dim3 threads(NTHREAD, 1, 1);
	int sharedMemSize = NTHREAD * sizeof(Particle);
	// pot_kernel <<<grid, threads, sharedMemSize >>> (n, ptcl_d, phi_d);
	pot_kernel <<<grid, threads, sharedMemSize >>> (n, istart, ptcl, phi);

	// hipMemcpy(phi_h, phi_d, n * sizeof(float2), hipMemcpyDeviceToHost);
	phi.dtoh(ni);
	for(int i=0; i<ni; i++){
		// pot[i] = (double)phi_h[i].x + (double)phi_h[i].y;
		pot[i] = (double)phi[i].x + (double)phi[i].y;
	}

	phi.free();
	ptcl.free();
	double t1 = get_wtime();
#ifdef PROFILE
	fprintf(stderr, "rank %d  Ni: %d NTOT: %d  gpupot: %f sec\n",*rank,ni,n,t1 - t0);
#endif
}

extern "C"{
	void gpupot_(
            int *irank,
            int *istart,
            int *ni,
			int *n,
			double m[],
			double x[][3],
			double pot[]){
      gpupot(irank, *istart, *ni, *n, m, x, pot);
	}
}

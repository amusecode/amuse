#include "hip/hip_runtime.h"
// Written by: Pablo Bena Llambay / Mark Gieles
// Calculates specific potential of a cluster (N1 = 0) or a cluster pair (0 < N1 < N)

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "mameclot.h"

#define BLOCKSIZE 256
	
//Computing the Potential on the Device
__global__ void compute_potential_gpu(float *m, 
	   float *x, float *y, float *z, float *phi, int N, int N1) {
  int i,j; 
  float rijx, rijy, rijz;
  float xi, yi, zi;
  float potential;	
  i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < (N1 == 0 ? N : N1)) 
  {
    xi = x[i];
    yi = y[i];
    zi = z[i];

    for (j = (N1 == 0 ? 0 : N1); j < N; j++) 
    {
      rijx = xi - x[j];
      rijy = yi - y[j];
      rijz = zi - z[j];
     
      if (i!=j)  
         potential -= m[j]/sqrt(rijx*rijx + rijy*rijy + rijz*rijz);
    }
   phi[i] = potential;
   }   
}


extern "C" void calculate_potential(float *m, float *x, float *y, float *z,
        float *phi, int N, int N1) 
{
  float *m_d,*x_d,*y_d,*z_d,*phi_d; // Device variables!

  //Allocating memory on the Device
  hipMalloc(&m_d  , sizeof(float)*N); 
  hipMalloc(&x_d  , sizeof(float)*N); 
  hipMalloc(&y_d  , sizeof(float)*N);
  hipMalloc(&z_d  , sizeof(float)*N); 
  hipMalloc(&phi_d, sizeof(float)*N);

  hipMemcpy(m_d,m    , sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device
  hipMemcpy(x_d,x    , sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device
  hipMemcpy(y_d,y    , sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device
  hipMemcpy(z_d,z    , sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device
  hipMemcpy(phi_d,phi, sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device

  compute_potential_gpu <<<((N+BLOCKSIZE-1))/BLOCKSIZE,BLOCKSIZE >>>(m_d,x_d, y_d, z_d, phi_d,N,N1);
  hipMemcpy(phi,phi_d, sizeof(float)*N, hipMemcpyDeviceToHost); // Host -> Device
    
  //Freeing memory
  hipFree(m_d);
  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);
  hipFree(phi_d);
  
  return;
}


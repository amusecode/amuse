#include "hip/hip_runtime.h"
#include "support_kernels.cu"
#include <stdio.h>


//////////////////////////////
//////////////////////////////
//////////////////////////////

//Helper functions for leaf-nodes
__device__ void compute_monopole(float2 &mass, float2 &posx, 
                                 float2 &posy, float2 &posz,
                                 float4 pos)
{
  mass = ds_accumulate(mass, pos.w);
  posx = ds_accumulate(posx, pos.w*pos.x);
  posy = ds_accumulate(posy, pos.w*pos.y);
  posz = ds_accumulate(posz, pos.w*pos.z);
}

__device__ void compute_quadropole(float2 &oct_q11, float2 &oct_q22, float2 &oct_q33, 
                                   float2 &oct_q12, float2 &oct_q13, float2 &oct_q23, 
                                   float4 pos)
{  
  oct_q11 = ds_accumulate(oct_q11, pos.w * pos.x*pos.x);
  oct_q22 = ds_accumulate(oct_q22, pos.w * pos.y*pos.y);
  oct_q33 = ds_accumulate(oct_q33, pos.w * pos.z*pos.z);
  oct_q12 = ds_accumulate(oct_q12, pos.w * pos.x*pos.y);
  oct_q13 = ds_accumulate(oct_q13, pos.w * pos.y*pos.z);
  oct_q23 = ds_accumulate(oct_q23, pos.w * pos.z*pos.x);

}

__device__ void compute_bounds(float3 &r_min, float3 &r_max, 
                               float4 pos)
{
  r_min.x = fminf(r_min.x, pos.x);
  r_min.y = fminf(r_min.y, pos.y);
  r_min.z = fminf(r_min.z, pos.z);

  r_max.x = fmaxf(r_max.x, pos.x);
  r_max.y = fmaxf(r_max.y, pos.y);
  r_max.z = fmaxf(r_max.z, pos.z);
}

//Non-leaf node helper functions
__device__ void compute_quadropole_node(float2 &oct_q11, float2 &oct_q22, float2 &oct_q33, 
                                   float2 &oct_q12, float2 &oct_q13, float2 &oct_q23, 
                                   float4 Q0, float4 Q1)
{  
  oct_q11 = ds_accumulate(oct_q11, Q0.x);
  oct_q22 = ds_accumulate(oct_q22, Q0.y);
  oct_q33 = ds_accumulate(oct_q33, Q0.z);
  oct_q12 = ds_accumulate(oct_q12, Q1.x);
  oct_q13 = ds_accumulate(oct_q13, Q1.y);
  oct_q23 = ds_accumulate(oct_q23, Q1.z);
}

__device__ void compute_bounds_node(float3 &r_min, float3 &r_max, 
                                    float4 node_min, float4 node_max)
{
  r_min.x = fminf(r_min.x, node_min.x);
  r_min.y = fminf(r_min.y, node_min.y);
  r_min.z = fminf(r_min.z, node_min.z);

  r_max.x = fmaxf(r_max.x, node_max.x);
  r_max.y = fmaxf(r_max.y, node_max.y);
  r_max.z = fmaxf(r_max.z, node_max.z);
}


extern "C" __global__ void compute_leaf(const int n_leafs,  
                                            uint *leafsIdxs,
                                            uint2 *node_bodies,
                                            real4 *body_pos,
                                            real4 *multipole,
                                            real4 *nodeLowerBounds,
                                            real4 *nodeUpperBounds,
                                            float3 *lowerBounds,
                                            float3 *upperBounds,
                                            real4  *body_vel) {

  const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
  const uint tid = threadIdx.x;
  const uint id  = bid * blockDim.x + tid;


  volatile __shared__ float3 shmem[256];
  volatile float3 *sh_rmin = (float3*)&shmem [ 0]; 
  volatile float3 *sh_rmax = (float3*)&shmem[128]; 

  //Set shared memory to defaults and return
  if (id >= n_leafs)
  {
    sh_rmin[tid].x = +1e10f; sh_rmin[tid].y = +1e10f; sh_rmin[tid].z = +1e10f;
    sh_rmax[tid].x = -1e10f; sh_rmax[tid].y = -1e10f; sh_rmax[tid].z = -1e10f;    
    return;
  }


  //Since nodes are intermixes with non-leafs in the node_bodies array
  //we get a leaf-id from the leafsIdxs array
  int nodeID = leafsIdxs[id]; 
  
  const uint2 bij          =  node_bodies[nodeID];
  const uint firstChild    =  bij.x & ILEVELMASK;
  const uint lastChild     =  bij.y;  //TODO maybe have to increase it by 1

  //Variables holding properties and intermediate answers
  float4 p;
  float2 mass, posx, posy, posz;
  mass = posx = posy = posz = (float2){0.0f, 0.0f};
  float2 oct_q11, oct_q22, oct_q33;
  float2 oct_q12, oct_q13, oct_q23;
  oct_q11 = oct_q22 = oct_q33 = (float2){0.0f, 0.0f};
  oct_q12 = oct_q13 = oct_q23 = (float2){0.0f, 0.0f};
  float3 r_min, r_max;
  r_min = (float3){+1e10f, +1e10f, +1e10f}; 
  r_max = (float3){-1e10f, -1e10f, -1e10f}; 

  //Loop over the children=>particles=>bodys
  //unroll increases register usage #pragma unroll 16
  float maxEps = 0.0f;
  int count=0;
  for(int i=firstChild; i < lastChild; i++)
  {
    p      = body_pos[i];
    maxEps = fmaxf(body_vel[i].w, maxEps);      //Determine the max softening within this leaf
    count++;
    compute_monopole(mass, posx, posy, posz, p);
    compute_quadropole(oct_q11, oct_q22, oct_q33, oct_q12, oct_q13, oct_q23, p);
    compute_bounds(r_min, r_max, p);   
  }

  float4 mon = {ds_regularise(posx).x, ds_regularise(posy).x, ds_regularise(posz).x, ds_regularise(mass).x};  

  float im = 1.0f/mon.w;
  mon.x *= im;    
  mon.y *= im;
  mon.z *= im;

  float4 Q0, Q1;
  Q0   = (float4){ds_regularise(oct_q11).x, ds_regularise(oct_q22).x, ds_regularise(oct_q33).x, maxEps};
  Q1   = (float4){ds_regularise(oct_q12).x, ds_regularise(oct_q13).x, ds_regularise(oct_q23).x, 0.0f};

  //Store the node properties
  multipole[3*nodeID + 0] = mon;       //Monopole
  multipole[3*nodeID + 1] = Q0;        //Quadropole
  multipole[3*nodeID + 2] = Q1;        //Quadropole

  //TODO why is this required again?
  nodeLowerBounds[nodeID] = (float4){r_min.x, r_min.y, r_min.z, 0.0f};
  nodeUpperBounds[nodeID] = (float4){r_max.x, r_max.y, r_max.z, 1.0f};  //4th parameter is set to 1 to indicate this is a leaf

  //Global domain boundaries using reduction
  sh_rmin[tid].x = r_min.x; sh_rmin[tid].y = r_min.y; sh_rmin[tid].z = r_min.z;
  sh_rmax[tid].x = r_max.x; sh_rmax[tid].y = r_max.y; sh_rmax[tid].z = r_max.z;
  __syncthreads();

  //Reduction of the global boundaries of the system
  if(blockDim.x >= 128) if (tid < 64) {sh_MinMax(tid, tid + 64, &r_min, &r_max, sh_rmin, sh_rmax);} __syncthreads();
  if (tid < 32) 
  {
    sh_MinMax(tid, tid + 32, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid + 16, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  8, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  4, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  2, &r_min, &r_max, sh_rmin,sh_rmax);
    sh_MinMax(tid, tid +  1, &r_min, &r_max, sh_rmin,sh_rmax);
  }
  __syncthreads();

  //Store the results
  if(tid == 0)
  {
    //Compiler doesnt allow: volatile float3 = float3
    lowerBounds[bid].x = sh_rmin[0].x; lowerBounds[bid].y = sh_rmin[0].y; lowerBounds[bid].z = sh_rmin[0].z;
    upperBounds[bid].x = sh_rmax[0].x; upperBounds[bid].y = sh_rmax[0].y; upperBounds[bid].z = sh_rmax[0].z;
  }

  return;
}


//Function goes level by level (starting from deepest) and computes
//the properties of the non-leaf nodes
extern "C" __global__ void compute_non_leaf(const int curLevel,         //Level for which we calc                                            
                                            uint  *leafsIdxs,           //Conversion of ids
                                            uint  *node_level_list,     //Contains the start nodes of each lvl
                                            uint  *n_children,          //Reference from node to first child and number of childs
                                            real4 *multipole,
                                            real4 *nodeLowerBounds,
                                            real4 *nodeUpperBounds){

  const int bid =  blockIdx.y *  gridDim.x +  blockIdx.x;
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;

  const int idx = bid * (blockDim.x * blockDim.y) + tid;
  
  const int endNode   = node_level_list[curLevel];
  const int startNode = node_level_list[curLevel-1];

  
  if(idx >= (endNode-startNode))     return;

  const int nodeID = leafsIdxs[idx + startNode];

  //Get the children info
  const uint firstChild = n_children[nodeID] & 0x0FFFFFFF;            //TODO make this name/define?
  const uint nChildren  = ((n_children[nodeID]  & 0xF0000000) >> 28); //TODO make this name/define?

  //Variables
  float2 mass, posx, posy, posz;
  mass = posx = posy = posz = (float2){0.0f, 0.0f};
  float2 oct_q11, oct_q22, oct_q33;
  float2 oct_q12, oct_q13, oct_q23;
  oct_q11 = oct_q22 = oct_q33 = (float2){0.0f, 0.0f};
  oct_q12 = oct_q13 = oct_q23 = (float2){0.0f, 0.0f};
  float3 r_min, r_max;
  r_min = (float3){+1e10f, +1e10f, +1e10f}; 
  r_max = (float3){-1e10f, -1e10f, -1e10f}; 

  float maxEps = 0;
  //Process the children (1 to 8)
  for(int i=firstChild; i < firstChild+nChildren; i++)
  {
    //Gogo process this data!
    float4 tmon = multipole[3*i + 0];

    maxEps = fmaxf(multipole[3*i + 1].w, maxEps);

    compute_monopole(mass, posx, posy, posz, tmon);
    compute_quadropole_node(oct_q11, oct_q22, oct_q33, oct_q12, oct_q13, oct_q23, 
                            multipole[3*i + 1], multipole[3*i + 2]);
    compute_bounds_node(r_min, r_max, nodeLowerBounds[i], nodeUpperBounds[i]);
  }

  //Save the bounds 
  nodeLowerBounds[nodeID] = (float4){r_min.x, r_min.y, r_min.z, 0.0f};
  nodeUpperBounds[nodeID] = (float4){r_max.x, r_max.y, r_max.z, 0.0f}; //4th is set to 0 to indicate a non-leaf

  //Regularize and store the results
  float4 mon = {ds_regularise(posx).x, ds_regularise(posy).x, ds_regularise(posz).x, ds_regularise(mass).x};  
  float im = 1.0f/mon.w;
  mon.x *= im;    
  mon.y *= im;
  mon.z *= im;

  float4 Q0, Q1;
  Q0   = (float4){ds_regularise(oct_q11).x, ds_regularise(oct_q22).x, ds_regularise(oct_q33).x, maxEps};
  Q1   = (float4){ds_regularise(oct_q12).x, ds_regularise(oct_q13).x, ds_regularise(oct_q23).x, 0.0f};

  multipole[3*nodeID + 0] = mon;        //Monopole
  multipole[3*nodeID + 1] = Q0;         //Quadropole1
  multipole[3*nodeID + 2] = Q1;         //Quadropole2

  return;
}
extern "C" __global__ void compute_scaling(const int node_count,                                           
                                           real4 corner,
                                           real4 *multipole,
                                           real4 *nodeLowerBounds,
                                           real4 *nodeUpperBounds,
                                           uint  *n_children, 
                                           uint4 *node_data,
                                           float theta,
                                           real4 *boxSizeInfo,
                                           real4 *boxCenterInfo){


  const int bid =  blockIdx.y *  gridDim.x +  blockIdx.x;
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;

  const int idx = bid * (blockDim.x * blockDim.y) + tid;

  if(idx >= node_count)     return;

  float4 mon, Q0, Q1;

  mon = multipole[3*idx + 0];        //Monopole
  Q0  = multipole[3*idx + 1];        //Quadropole1
  Q1  = multipole[3*idx + 2];        //Quadropole2
  
  //Scale the quadropole
  float im = 1.0f / mon.w;
  Q0.x = Q0.x*im - mon.x*mon.x;
  Q0.y = Q0.y*im - mon.y*mon.y;
  Q0.z = Q0.z*im - mon.z*mon.z;
  Q1.x = Q1.x*im - mon.x*mon.y;
  Q1.y = Q1.y*im - mon.y*mon.z;
  Q1.z = Q1.z*im - mon.x*mon.z;

  //Switch the y and z parameter
  real temp = Q1.y;
  Q1.y = Q1.z; Q1.z = temp;

  multipole[3*idx + 1] = Q0;        //Quadropole1
  multipole[3*idx + 2] = Q1;        //Quadropole2


  float4 r_min, r_max;
  r_min = nodeLowerBounds[idx];
  r_max = nodeUpperBounds[idx];





  float3 boxCenter;
  boxCenter.x = 0.5*(r_min.x + r_max.x);
  boxCenter.y = 0.5*(r_min.y + r_max.y);
  boxCenter.z = 0.5*(r_min.z + r_max.z);

  float3 boxSize = (float3){fmaxf(fabs(boxCenter.x-r_min.x), fabs(boxCenter.x-r_max.x)),
                          fmaxf(fabs(boxCenter.y-r_min.y), fabs(boxCenter.y-r_max.y)),
                          fmaxf(fabs(boxCenter.z-r_min.z), fabs(boxCenter.z-r_max.z))};

  //Calculate distance between center of the box and the center of mass
  float3 s3     = (float3){(boxCenter.x - mon.x), (boxCenter.y - mon.y), (boxCenter.z - mon.z)};
  double s      = sqrt((s3.x*s3.x) + (s3.y*s3.y) + (s3.z*s3.z));

  //Length of the box, note times 2 since we only computed half the distance before
  float l = 2*fmaxf(boxSize.x, fmaxf(boxSize.y, boxSize.z));

  //Extra check, shouldnt be necessary
//  if(l < 0.000001)
 //   l = 0.000001;

  //Store the box size and opening criteria
  boxSizeInfo[idx].x = boxSize.x;
  boxSizeInfo[idx].y = boxSize.y;
  boxSizeInfo[idx].z = boxSize.z;
  boxSizeInfo[idx].w = __int_as_float(n_children[idx]);

  boxCenterInfo[idx].x = boxCenter.x;
  boxCenterInfo[idx].y = boxCenter.y;
  boxCenterInfo[idx].z = boxCenter.z;

  #ifdef IMPBH
    float cellOp = (l/theta) + s;
  #else
    //Minimum distance method
    float cellOp = (l/theta); 
  #endif
    
  cellOp = cellOp*cellOp;

  if(r_max.w > 0)
  {
    cellOp = -cellOp;       //This is a leaf node
  }

  boxCenterInfo[idx].w = cellOp;

 /* //Determine the size of the node based on the center of mass and the bounds of the node                                                                   
  float3 size3  = (float3){fmaxf(fabs(mon.x-r_min.x), fabs(mon.x-r_max.x)),                                                                                  
                          fmaxf(fabs(mon.y-r_min.y), fabs(mon.y-r_max.y)),                                                                                  
                          fmaxf(fabs(mon.z-r_min.z), fabs(mon.z-r_max.z))};                                                                                 
  float size    = fmaxf(size3.x, fmaxf(size3.y, size3.z));                                                                                                     

  //Box properties
  float3 boxCenter;
  boxCenter.x = 0.5*(r_min.x + r_max.x);
  boxCenter.y = 0.5*(r_min.y + r_max.y);
  boxCenter.z = 0.5*(r_min.z + r_max.z);

  float3 boxSize3 = (float3){fmaxf(fabs(boxCenter.x-r_min.x), fabs(boxCenter.x-r_max.x)),                                                                                  
                          fmaxf(fabs(boxCenter.y-r_min.y), fabs(boxCenter.y-r_max.y)),                                                                                  
                          fmaxf(fabs(boxCenter.z-r_min.z), fabs(boxCenter.z-r_max.z))};                    
  
  //Calculate distance between center of the box and the center of mass
//   float3 s3     = (float3){(boxCenter.x - mon.x), (boxCenter.y - mon.y), (boxCenter.z - mon.z)};    
//   double s      = sqrt((s3.x*s3.x) + (s3.y*s3.y) + (s3.z*s3.z));

  //BH: l/theta + s < d
  float l = fmaxf(boxSize3.x, fmaxf(boxSize3.y, boxSize3.z));
  
//   float cellOp = (l/theta) + s;
  float cellOp;
  cellOp = cellOp*cellOp;

  //Store the box size and opening criteria
//   cellOpening[idx].x = boxSize.x;
//   cellOpening[idx].y = boxSize.y;
//   cellOpening[idx].z = boxSize.z;
//   cellOpening[idx].w = cellOp;

  //Store the properties of the node
  boxCenterInfo[idx].x = boxCenter.x;
  boxCenterInfo[idx].y = boxCenter.y;
  boxCenterInfo[idx].z = boxCenter.z;
  boxCenterInfo[idx].w = __int_as_float(n_children[idx]);

  boxSize[idx].x = boxSize3.x;
  boxSize[idx].y = boxSize3.y;
  boxSize[idx].z = boxSize3.z;
  if(r_max.w > 0)  
    boxSize[idx].w = -1; //Leaf node
  else
     boxSize[idx].w = 1; // non-leaf node
 

  //Calculate the key 
  int4 crd;

  float domain_fac = corner.w; 

  float idomain_fac = 1.0f / domain_fac;
  crd.x = (int)((mon.x - corner.x) * idomain_fac + 0.5);
  crd.y = (int)((mon.y - corner.y) * idomain_fac + 0.5);
  crd.z = (int)((mon.z - corner.z) * idomain_fac + 0.5);
  uint2 key = get_key(crd);
  
  //Use the key to calculate back the position
  float3 pos;
  pos.x = crd.x*domain_fac + corner.x;
  pos.y = crd.y*domain_fac + corner.y;
  pos.z = crd.z*domain_fac + corner.z;

  //Adjust size based on the  key-based position of the node
  float ds = fmax(fabs(pos.x - mon.x), max(fabs(pos.y - mon.y), fabs(pos.z - mon.z)));  
  temp = size;
  size += ds;

  #ifdef IMPBH
  //Box size, max size for now
  size = l;
  if(l < 0.000001)
    size = 0.000001;
  
  #endif

  if(r_max.w > 0)
  {
    size = -size;       //This is a leaf node
  }

  //nchildren contains the node to node references
  //we also need to use node_bodies to get the 
  //leaf-particle references
  node_data[idx] = (uint4){key.x, key.y, 
                          __float_as_int(size), 
                          n_children[idx]};*/

//  r_min.w = size;
//  nodeLowerBounds[idx] = r_min;

  return;
}

//Modify the references to the fist body and the number of bodys
//for the leafs
//Also copy the node_data to the group data
extern "C" __global__ void copyNodeDataToGroupData(const int n_groups,         
                                                   const int n_nodes,                                                 
                                                   uint4 *node_data,
                                                   uint4 *group_data,
                                                   uint2 *node_bodies,
                                                   int   *group_list,
                                                   real4 *boxCenterInfo,
                                                   real4 *boxSizeInfo,
                                                   real4 *groupCenterInfo,
                                                   real4 *groupSizeInfo){
  const int bid =  blockIdx.y *  gridDim.x +  blockIdx.x;
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;

  const int idx = bid * (blockDim.x * blockDim.y) + tid;

  if(idx >= n_nodes)     return;

  //Copy the data and change the children data
  //Instead of pointing to child nodes we want it to point to
  //particles
//   uint4 nodeData = node_data[idx];
//   bool leaf =  __int_as_float(nodeData.z) <= 0;

  float temp = boxCenterInfo[idx].w;
  bool leaf = temp <= 0;

  //uint2 bij2     = node_bodies[idx];
  //uint pfirst2   = bij2.x & ILEVELMASK;
  //uint nchild2   = bij2.y - pfirst2;

  
  //Change the indirections of the leaf nodes so they point to
  //the particle data
  if(leaf)
  {
    uint2 bij     = node_bodies[idx];
    uint pfirst   = bij.x & ILEVELMASK;
    uint nchild   = bij.y - pfirst;

    pfirst = pfirst | ((nchild-1) << LEAFBIT);
    boxSizeInfo[idx].w = __int_as_float(pfirst);
  }

  //Now fill in the group data
  if(idx >= n_groups)     return;

  int nodeID         = group_list[idx];
  real4 nodeData     = boxSizeInfo[nodeID];

  uint2 bij     = node_bodies[nodeID];
  int pfirst    = bij.x & ILEVELMASK;
  int nchild    = bij.y - pfirst;

  pfirst = pfirst | (nchild-1) << CRITBIT;
  nodeData.w = __int_as_float(pfirst);

  groupSizeInfo[idx]   = nodeData;  
  groupCenterInfo[idx] = boxCenterInfo[nodeID];
}


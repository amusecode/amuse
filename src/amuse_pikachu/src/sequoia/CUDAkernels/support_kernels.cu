#include "hip/hip_runtime.h"
//Definitions

#include "node_specs.h"

#include <stdio.h>

typedef unsigned int uint;

typedef float real;
typedef float4 real4;

/*

int3 gives problems with memory copies
therefor im using int4
Wrong type for attribute nocapture                                                                     
void (i64, i8*, <unrecognized-type>, i32)* @llvm.memcpy.i64                                            
Argument value does not match function argument type!                                                  
void %2                                                                                                
 <unrecognized-type>Broken module found, compilation aborted!                                          
Aborted   

typedef struct int3
{
  int x,y,z;
} int3;
*/


__device__ int undilate3(uint2 key) {
  int x, value = 0;
  
  key.x = key.x & 0x09249249;
  key.y = key.y & 0x09249249;
  
  // undilate first 10 bits

  x = key.y & 0x3FFFF;
  x = ((x <<  4) + (x << 2) + x) & 0x0E070381;
  x = ((x << 12) + (x << 6) + x) & 0x0FF80001;
  x = ((x << 18) + x) & 0x0FFC0000;
  value = value | (x >> 18);
  
  x = (key.y >> 18) & 0x3FFFF;
  x = ((x <<  4) + (x << 2) + x) & 0x0E070381;
  x = ((x << 12) + (x << 6) + x) & 0x0FF80001;
  x = ((x << 18) + x) & 0x0FFC0000;
  value = value | (x >> 12);
  

  // undilate second 10 bits

  x = key.x & 0x3FFFF;
  x = ((x <<  4) + (x << 2) + x) & 0x0E070381;
  x = ((x << 12) + (x << 6) + x) & 0x0FF80001;
  x = ((x << 18) + x) & 0x0FFC0000;
  value = value | ((x >> 18) << 10);
  
  x = (key.x >> 18) & 0x3FFFF;
  x = ((x <<  4) + (x << 2) + x) & 0x0E070381;
  x = ((x << 12) + (x << 6) + x) & 0x0FF80001;
  x = ((x << 18) + x) & 0x0FFC0000;
  value = value | ((x >> 12) << 10);
  
  return value;
}


__device__ uint2 dilate3(int value) {
  unsigned int x;
  uint2 key;
  
  // dilate first 10 bits

  x = value & 0x03FF;
  x = ((x << 16) + x) & 0xFF0000FF;
  x = ((x <<  8) + x) & 0x0F00F00F;
  x = ((x <<  4) + x) & 0xC30C30C3;
  x = ((x <<  2) + x) & 0x49249249;
  key.y = x;

  // dilate second 10 bits

  x = (value >> 10) & 0x03FF;
  x = ((x << 16) + x) & 0xFF0000FF;
  x = ((x <<  8) + x) & 0x0F00F00F;
  x = ((x <<  4) + x) & 0xC30C30C3;
  x = ((x <<  2) + x) & 0x49249249;
  key.x = x;

  return key;
} 

#if 0
__device__ uint2 get_key(int4 crd) {
  uint2 key, key1;
  key  = dilate3(crd.x);

  key1 = dilate3(crd.y);
  key.x = key.x | (key1.x << 1);
  key.y = key.y | (key1.y << 1);

  key1 = dilate3(crd.z);
  key.x = key.x | (key1.x << 2);
  key.y = key.y | (key1.y << 2);

  return key;
}

#else

#if 0
__device__ uint4 get_key(int4 crd)
{
  const int bits = 20;  //20 to make it same number as morton order
  int i,xi, yi, zi;
  int mask;
  int key;
    
  //0= 000, 1=001, 2=011, 3=010, 4=110, 5=111, 6=101, 7=100
  //000=0=0, 001=1=1, 011=3=2, 010=2=3, 110=6=4, 111=7=5, 101=5=6, 100=4=7
  const int C[8] = {0, 1, 7, 6, 3, 2, 4, 5};
    
  int temp;
    
  mask = 1 << (bits - 1);
  key  = 0;

  uint4 key_new;
    
  for(i = 0; i < bits; i++, mask >>= 1)
  {
    xi = (crd.x & mask) ? 1 : 0;
    yi = (crd.y & mask) ? 1 : 0;
    zi = (crd.z & mask) ? 1 : 0;        

    int index = (xi << 2) + (yi << 1) + zi;

      
    if(index == 0)
    {
      temp = crd.z; crd.z = crd.y; crd.y = temp;
    }
    else  if(index == 1 || index == 5)
    {
      temp = crd.x; crd.x = crd.y; crd.y = temp;
    }
    else  if(index == 4 || index == 6)
    {
      crd.x = (crd.x) ^ (-1);
      crd.z = (crd.z) ^ (-1);
    }
    else  if(index == 7 || index == 3)
    {
      temp = (crd.x) ^ (-1);         
      crd.x = (crd.y) ^ (-1);
      crd.y = temp;
    }
    else
    {
      temp = (crd.z) ^ (-1);         
      crd.z = (crd.y) ^ (-1);
      crd.y = temp;          
    }   

    key = (key << 3) + C[index];

    if(i == 9)
    {
      key_new.x = key;
      key = 0;
    }
  } //end for

   key_new.y = key;

  return key_new;
}
#else


__device__ uint4 get_key(int4 crd)
{
  const int bits = 30;  //20 to make it same number as morton order
  int i,xi, yi, zi;
  int mask;
  int key;
    
  //0= 000, 1=001, 2=011, 3=010, 4=110, 5=111, 6=101, 7=100
  //000=0=0, 001=1=1, 011=3=2, 010=2=3, 110=6=4, 111=7=5, 101=5=6, 100=4=7
  const int C[8] = {0, 1, 7, 6, 3, 2, 4, 5};
    
  int temp;
    
  mask = 1 << (bits - 1);
  key  = 0;

  uint4 key_new;
    
  for(i = 0; i < bits; i++, mask >>= 1)
  {
    xi = (crd.x & mask) ? 1 : 0;
    yi = (crd.y & mask) ? 1 : 0;
    zi = (crd.z & mask) ? 1 : 0;        

    int index = (xi << 2) + (yi << 1) + zi;
      
    if(index == 0)
    {
      temp = crd.z; crd.z = crd.y; crd.y = temp;
    }
    else  if(index == 1 || index == 5)
    {
      temp = crd.x; crd.x = crd.y; crd.y = temp;
    }
    else  if(index == 4 || index == 6)
    {
      crd.x = (crd.x) ^ (-1);
      crd.z = (crd.z) ^ (-1);
    }
    else  if(index == 7 || index == 3)
    {
      temp = (crd.x) ^ (-1);         
      crd.x = (crd.y) ^ (-1);
      crd.y = temp;
    }
    else
    {
      temp = (crd.z) ^ (-1);         
      crd.z = (crd.y) ^ (-1);
      crd.y = temp;          
    }   

    key = (key << 3) + C[index];

// Hier gebleven, zorgen dat juiste bits op juiste plek komen
    if(i == 19)
    {
      key_new.y = key;
      key = 0;
    }
    if(i == 9)
    {
      key_new.x = key;
      key = 0;
    }
  } //end for

   key_new.z = key;

  return key_new;
}

#endif


#if 0
__device__ uint2 get_key(int4 crd)
{
  const int bits = 20;  //20 to make it same number as morton order
  int i,xi, yi, zi;
  int mask;
  long key;
    
  //0= 000, 1=001, 2=011, 3=010, 4=110, 5=111, 6=101, 7=100
  //000=0=0, 001=1=1, 011=3=2, 010=2=3, 110=6=4, 111=7=5, 101=5=6, 100=4=7
  const int C[8] = {0, 1, 7, 6, 3, 2, 4, 5};
    
  int temp;
    
  mask = 1 << (bits - 1);
  key  = 0;
    
  for(i = 0; i < bits; i++, mask >>= 1)
  {
    xi = (crd.x & mask) ? 1 : 0;
    yi = (crd.y & mask) ? 1 : 0;
    zi = (crd.z & mask) ? 1 : 0;        

    int index = (xi << 2) + (yi << 1) + zi;

      
    if(index == 0)
    {
      temp = crd.z; crd.z = crd.y; crd.y = temp;
    }
    else  if(index == 1 || index == 5)
    {
      temp = crd.x; crd.x = crd.y; crd.y = temp;
    }
    else  if(index == 4 || index == 6)
    {
      crd.x = (crd.x) ^ (-1);
      crd.z = (crd.z) ^ (-1);
    }
    else  if(index == 7 || index == 3)
    {
      temp = (crd.x) ^ (-1);         
      crd.x = (crd.y) ^ (-1);
      crd.y = temp;
    }
    else
    {
      temp = (crd.z) ^ (-1);         
      crd.z = (crd.y) ^ (-1);
      crd.y = temp;          
    }   

    key = (key << 3) + C[index];
  }
  

  uint2 key_new;
//   key_new.x = key & 0xFFFFFFFF;
//   key_new.y = (key >> 32) & 0xFFFFFFFF;
  key_new.y = key         & 0xFFFFFFFF;
  key_new.x = (key >> 32) & 0xFFFFFFFF;


  return key_new;
}
#endif

#endif

/*
__device__ uint2 get_mask(int level) {
  int mask_levels = 3*max(MAXLEVELS - level, 0);
  uint2 mask = {0x3FFFFFFF, 0xFFFFFFFF};
  
  if (mask_levels > 30) {
    mask.y = 0;
    mask.x = (mask.x >> (mask_levels - 30)) << (mask_levels - 30);
  } else {
    mask.y = (mask.y >> mask_levels) << mask_levels;
  }
  
  return mask;
}*/



__device__ uint4 get_mask(int level) {
  int mask_levels = 3*max(MAXLEVELS - level, 0);
  uint4 mask = {0x3FFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF,0xFFFFFFFF};

  if (mask_levels > 60)
  {
    mask.z = 0;
    mask.y = 0;
    mask.x = (mask.x >> (mask_levels - 60)) << (mask_levels - 60);
  }
  else if (mask_levels > 30) {
    mask.z = 0;
    mask.y = (mask.y >> (mask_levels - 30)) << (mask_levels - 30);
  } else {
    mask.z = (mask.z >> mask_levels) << mask_levels;
  }

// if(threadIdx.x == 0 && blockIdx.x == 0)
// {
//   printf("ON DEV TEST: lvl: %d mlvl: %d x: %d y: %d z: %d \n", level, mask_levels, mask.x, mask.y, mask.z);
// }
//   
  return mask;
}

/*
__device__ uint2 get_imask(uint2 mask) {
  return (uint2){0x3FFFFFFF ^ mask.x, 0xFFFFFFFF ^ mask.y};
}*/

__device__ uint4 get_imask(uint4 mask) {
  return (uint4){0x3FFFFFFF ^ mask.x, 0xFFFFFFFF ^ mask.y, 0xFFFFFFFF ^ mask.z, 0};
}



__device__ int4 get_crd(uint2 key) {
  int4 crd;

  crd.x = undilate3(key);
  crd.y = undilate3((uint2){key.x >> 1, key.y >> 1});
  crd.z = undilate3((uint2){key.x >> 2, key.y >> 2});
  
  return crd;
}

__device__ int cmp_uint2(uint2 a, uint2 b) {
  if      (a.x < b.x) return -1;
  else if (a.x > b.x) return +1;
  else {
    if       (a.y < b.y) return -1;
    else  if (a.y > b.y) return +1;
    return 0;
  }  
}

__device__ int cmp_uint4(uint4 a, uint4 b) {
  if      (a.x < b.x) return -1;
  else if (a.x > b.x) return +1;
  else {
    if       (a.y < b.y) return -1;
    else  if (a.y > b.y) return +1;
    else {
      if       (a.z < b.z) return -1;
      else  if (a.z > b.z) return +1;
      return 0;
    } //end z    
  }  //end y
} //end x, function


#if 0
//Binary search of the key within certain bounds (cij.x, cij.y)
__device__ int find_key(uint2 key, uint2 cij, uint2 *keys) {
  int l = cij.x;
  int r = cij.y - 1;
  while (r - l > 1) {
    int m = (r + l) >> 1;
    int cmp = cmp_uint2(keys[m], key);
    if (cmp == -1) {
      l = m;
    } else { 
      r = m;
    }
  }
  if (cmp_uint2(keys[l], key) >= 0) return l;

  return r;
}
#endif

//Binary search of the key within certain bounds (cij.x, cij.y)
__device__ int find_key(uint4 key, uint2 cij, uint4 *keys) {
  int l = cij.x;
  int r = cij.y - 1;
  while (r - l > 1) {
    int m = (r + l) >> 1;
    int cmp = cmp_uint4(keys[m], key);
    if (cmp == -1) {
      l = m;
    } else { 
      r = m;
    }
  }
  if (cmp_uint4(keys[l], key) >= 0) return l;

  return r;
}



__device__ float2 ds_accumulate(float2 a, float b){
  float tmp = a.x + b;
  float del = (tmp - a.x) - b;
  a.x = tmp;
  a.y -= del;
  return a;
}
__device__ float2 ds_regularise(float2 a){
  float tmp = a.x + a.y;
  a.y -= (tmp - a.x);
  a.x = tmp;
  return a;
}

// __device__ void sh_MinMax(int i, int j, volatile float3 *sh_rmin, volatile  float3 *sh_rmax)
// {
//       sh_rmin[i].x  = fminf(sh_rmin[i].x, sh_rmin[j].x);
//       sh_rmin[i].y  = fminf(sh_rmin[i].y, sh_rmin[j].y);
//       sh_rmin[i].z  = fminf(sh_rmin[i].z, sh_rmin[j].z);
//       sh_rmax[i].x  = fmaxf(sh_rmax[i].x, sh_rmax[j].x);
//       sh_rmax[i].y  = fmaxf(sh_rmax[i].y, sh_rmax[j].y);
//       sh_rmax[i].z  = fmaxf(sh_rmax[i].z, sh_rmax[j].z);
// }
__device__ void sh_MinMax(int i, int j, float3 *r_min, float3 *r_max, volatile float3 *sh_rmin, volatile  float3 *sh_rmax)
{
  sh_rmin[i].x  = (*r_min).x = fminf((*r_min).x, sh_rmin[j].x);
  sh_rmin[i].y  = (*r_min).y = fminf((*r_min).y, sh_rmin[j].y);
  sh_rmin[i].z  = (*r_min).z = fminf((*r_min).z, sh_rmin[j].z);
  sh_rmax[i].x  = (*r_max).x = fmaxf((*r_max).x, sh_rmax[j].x);
  sh_rmax[i].y  = (*r_max).y = fmaxf((*r_max).y, sh_rmax[j].y);
  sh_rmax[i].z  = (*r_max).z = fmaxf((*r_max).z, sh_rmax[j].z);
}


__device__ void MinMaxPos(float4 pos, float4 &rmax, float4 &rmin)
{
      rmin.x  = fminf(pos.x, rmin.x);
      rmin.y  = fminf(pos.y, rmin.y);
      rmin.z  = fminf(pos.z, rmin.z);
      rmax.x  = fmaxf(pos.x, rmax.x); 
      rmax.y  = fmaxf(pos.y, rmax.y); 
      rmax.z  = fmaxf(pos.z, rmax.z); 
}


__device__ real4 get_pos(uint2 key, float size, float4 corner) {
  real4 pos;
  pos.w = size;
  
  int4 crd = get_crd(key);
  float domain_fac = corner.w;
  pos.x = crd.x*domain_fac + corner.x;
  pos.y = crd.y*domain_fac + corner.y;
  pos.z = crd.z*domain_fac + corner.z;

  return pos;
}

/***
**** --> prefix calculation via Horn(2005) data-parallel algoritm
***/
#define BTEST(x) (-(int)(x))
template<int DIM2>
__device__ int calc_prefix(int N, int* prefix_in, int tid) {
  int x, y = 0;

  const int DIM = 1 << DIM2;
  
  for (int p = 0; p < N; p += DIM) {
    int *prefix = &prefix_in[p];

    x = prefix[tid -  1]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  1); __syncthreads();
    x = prefix[tid -  2]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  2); __syncthreads();
    x = prefix[tid -  4]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  4); __syncthreads();
    x = prefix[tid -  8]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  8); __syncthreads();
    x = prefix[tid - 16]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 16); __syncthreads();
    if (DIM2 >= 6) {x = prefix[tid - 32]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 32); __syncthreads();}
    if (DIM2 >= 7) {x = prefix[tid - 64]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 64); __syncthreads();}
    if (DIM2 >= 8) {x = prefix[tid -128]; __syncthreads(); prefix[tid] += x & BTEST(tid >=128); __syncthreads();}
    

    prefix[tid] += y;
    __syncthreads();

    y = prefix[DIM-1];
    __syncthreads();
  }

  return y;
} 

template<int DIM2>
__device__ int calc_prefix(int* prefix, int tid, int value) {
  int  x;
  
  const int DIM = 1 << DIM2;

  prefix[tid] = value;
  __syncthreads();

#if 1
  x = prefix[tid -  1]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  1); __syncthreads();
  x = prefix[tid -  2]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  2); __syncthreads();
  x = prefix[tid -  4]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  4); __syncthreads();
  x = prefix[tid -  8]; __syncthreads(); prefix[tid] += x & BTEST(tid >=  8); __syncthreads();
  x = prefix[tid - 16]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 16); __syncthreads();
  if (DIM2 >= 6) {x = prefix[tid - 32]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 32); __syncthreads();}
  if (DIM2 >= 7) {x = prefix[tid - 64]; __syncthreads(); prefix[tid] += x & BTEST(tid >= 64); __syncthreads();}
  if (DIM2 >= 8) {x = prefix[tid -128]; __syncthreads(); prefix[tid] += x & BTEST(tid >=128); __syncthreads();}

  x = prefix[DIM - 1];
  __syncthreads();
  return x;
#else
  
  int offset = 0;
  int tid2 = tid << 1;

#pragma unroll
  for (int d = DIM >> 1; d > 0; d >>= 1) {
    __syncthreads();

    int iflag = BTEST(tid < d);
    int ai = (((tid2 + 1) << offset) - 1) & iflag;
    int bi = (((tid2 + 2) << offset) - 1) & iflag;
    
    prefix[bi] += prefix[ai] & iflag;
    offset++;
  }

  // clear the last element
  if (tid == 0) prefix[DIM - 1] = 0;

  // traverse down the tree building the scan in place
#pragma unroll
  for (int d = 1; d < DIM; d <<= 1) {
    offset--;
    __syncthreads();
    
    int iflag = BTEST(tid < d);
    int ai = (((tid2 + 1) << offset) - 1) & iflag;
    int bi = (((tid2 + 2) << offset) - 1) & iflag;
    
    int t       = prefix[ai];
    if (tid < d) {
      prefix[ai]  = (prefix[bi] & iflag) + (t & BTEST(tid >= d));
      prefix[bi] += t & iflag;
    }
  }
  __syncthreads();

  prefix[tid] += value;
  __syncthreads();
  
  x = prefix[DIM - 1];
  __syncthreads();
  return x;
#endif
}



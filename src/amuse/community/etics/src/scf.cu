#include "hip/hip_runtime.h"
/**
 * @file
 * @author  Yohai Meiron <ymeiron@pku.edu.cn>
 * @brief   Functions to calculate gravitational force using the SCF method.
 */
#include "common.hpp"
#include "mathaux.hpp"
#include "scf.hpp"
#include "ic.hpp"

#include <iostream>
using std::cout;
using std::cerr;
using std::endl;

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <mpi.h>

namespace etics {
    namespace scf {
        __constant__ Real RadCoeff[(NMAX+1)*(LMAX+1)];             /*!< Stores coefficients related to the G */
        __constant__ Real AngCoeff[(LMAX+1)*(LMAX+2)/2];           /*!< used blab bla222 */
        __constant__ Complex A[(NMAX+1)*(LMAX+1)*(LMAX+2)/2];
        __constant__ CacheStruct Cache;
                     Complex *PartialSum;

        Real RadCoeff_h[(NMAX+1)*(LMAX+1)];        /*!< Stores coefficients related to the G */
        Real AngCoeff_h[(LMAX+1)*(LMAX+2)/2];      /*!< used blab bla222 */
        Complex A_h[(NMAX+1)*(LMAX+1)*(LMAX+2)/2];
        CacheStruct Cache_h;
        Complex *PartialSum_h;

        int k3gs, k3bs, k4gs, k4bs;
    }
}

void etics::scf::InitializeCache(int N) { // not sure why it's a separate function, the instructions can be in etics::scf::Init()
    Cache_h.N = N;
    hipMalloc((void**)&Cache_h.xi,         N * sizeof(Real));
    hipMalloc((void**)&Cache_h.Phi0l,      N * sizeof(Real));
    hipMalloc((void**)&Cache_h.Wprev1,     N * sizeof(Real));
    hipMalloc((void**)&Cache_h.Wprev2,     N * sizeof(Real));
    hipMalloc((void**)&Cache_h.costheta,   N * sizeof(Real));
    hipMalloc((void**)&Cache_h.sintheta_I, N * sizeof(Real));
    hipMalloc((void**)&Cache_h.Exponent,   N * sizeof(Complex));
    hipMalloc((void**)&Cache_h.mass,       N * sizeof(Real));
}

void etics::scf::UpdateN(int N) {
    Cache_h.N = N;
    hipMemcpyToSymbol(HIP_SYMBOL(Cache), &Cache_h, sizeof(CacheStruct));
}

__global__ void etics::scf::LoadParticlesToCache(Particle *P, int N) { // formerly "Kernel1"
    int i = threadIdx.x + blockIdx.x *  blockDim.x;
    while (i < N) {
        vec3 Pos = P[i].pos;
        Real r = sqrt(Pos.x*Pos.x + Pos.y*Pos.y + Pos.z*Pos.z);
        Real xi = (r-1)/(r+1);
        Real costheta = Pos.z/r;
        Real sintheta_I = rsqrt(1-costheta*costheta);

        Cache.xi[i] = xi;
        Cache.Phi0l[i] = 0.5 * (1 - xi);
        Cache.costheta[i] = costheta;
        Cache.sintheta_I[i] = sintheta_I;

        Real Normal_I = rsqrt(Pos.x*Pos.x + Pos.y*Pos.y);
        Complex Exponent = make_Complex(Pos.x*Normal_I, -Pos.y*Normal_I);
        Cache.Exponent[i] = Exponent;
        Cache.mass[i] = P[i].m;

        i += blockDim.x * gridDim.x;
    }
}

__global__ void etics::scf::CalculatePhi0l(int l) { // formerly "Kernel2"
    int i = threadIdx.x + blockIdx.x *  blockDim.x;
    while (i < Cache.N) {
        Real xi = Cache.xi[i];
        Cache.Phi0l[i] *= 0.25*(1-xi*xi);

        i += blockDim.x * gridDim.x;
    }
}

__global__ void etics::scf::CalculateCoefficientsPartial(int n, int l, Complex *PartialSum) { // formerly "Kernel3"
    extern __shared__ Complex ReductionCache[]; // size is determined in kernel launch
    int tid = threadIdx.x;
    for (int m = 0; m <= l; m++) ReductionCache[m*blockDim.x+tid] = make_Complex(0, 0);
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    while (i < Cache.N) {
        Real xi = Cache.xi[i];
        Real Wnl;
        if (n == 0)      Wnl = 1;
        else if (n == 1) {Wnl = (4*l+3)*xi; Cache.Wprev2[i] = Wnl;}
        else if (n == 2) {Wnl = -(2*l+1.5)+( 8*l*(l+2) +7.5)*xi*xi; Cache.Wprev1[i] = Wnl;}
        else {
            Real Wprev1 = Cache.Wprev1[i];
            Wnl = (xi*(2*n+4*l+1)*Wprev1 - (n+4*l+1)*Cache.Wprev2[i])/(Real)n;
            if (n < NMAX) { // Writing is expensive, avoid if possible.
                Cache.Wprev2[i] = Wprev1;
                Cache.Wprev1[i] = Wnl;
            }
        }
        Real RadialPart = - Cache.mass[i] * SQRT_4_PI * Cache.Phi0l[i] * Wnl * RadCoeff[(LMAX+1)*n+l];
        Real costheta = Cache.costheta[i];
        Real Plm = Pl(l, costheta);
        ReductionCache[tid] = Complex_add(ReductionCache[tid], make_Complex(RadialPart * Plm * AngCoeff[(l+1)*l/2],0));
        if (l == 0) {i += blockDim.x * gridDim.x; continue;}

        //////////////////////////////// ugly fix
        if ((costheta < -0.999) || (costheta > +0.999)) {
            i += blockDim.x * gridDim.x;
            continue;
        }
        //////////////////////////////// ugly fix

        Real Plm_prev1 = Plm;
        Real sintheta_I = Cache.sintheta_I[i];
        Plm = (costheta*Plm - Pl(l-1, costheta))*l*sintheta_I;
        Complex Exponent = Cache.Exponent[i];
        Real tmp0 = RadialPart * Plm * AngCoeff[(l+1)*l/2+1];
        ReductionCache[blockDim.x+tid] = Complex_add(ReductionCache[blockDim.x+tid], make_Complex(tmp0 * Exponent.x, tmp0 * Exponent.y));

        if (l == 1) {i += blockDim.x * gridDim.x; continue;}

        Complex TorodialPart = Exponent;
        for (int m = 2; m <= l; m++) { // make sure no redundancy at the end of the loop
            Real Plm_prev2 = Plm_prev1;
            Plm_prev1 = Plm;
            Plm = - 2*(m-1)*costheta*sintheta_I*Plm_prev1 - (l+m-1)*(l-m+2)*Plm_prev2;
            TorodialPart = Complex_mul(TorodialPart, Exponent);
            tmp0 = RadialPart * Plm * AngCoeff[(l+1)*l/2+m];
            ReductionCache[m*blockDim.x+tid] = Complex_add(ReductionCache[m*blockDim.x+tid], make_Complex(tmp0 * TorodialPart.x, tmp0 * TorodialPart.y));
        }
        i += blockDim.x * gridDim.x;
    }
    __syncthreads();
    for (int m = 0; m <= l; m++) {
        i = blockDim.x/2;
        while (i != 0) {
            if (tid < i)
                ReductionCache[m*blockDim.x+tid] = Complex_add(ReductionCache[m*blockDim.x+tid], ReductionCache[m*blockDim.x+tid+i]);
            __syncthreads();
            i /= 2;
        }
        if (tid == 0)
            PartialSum[blockIdx.x*(l+1) + m] = ReductionCache[m*blockDim.x];
    }
}

void etics::scf::CalculateCoefficients(int n, int l, Complex *A_h) {
    int BaseAddress = n*(LMAX+1)*(LMAX+2)/2 + l*(l+1)/2;
    CalculateCoefficientsPartial<<<k3gs,k3bs,k3bs*sizeof(Complex)*(LMAX+1)>>>(n, l, PartialSum);
    hipMemcpy(PartialSum_h, PartialSum, k3gs*(l+1)*sizeof(Complex), hipMemcpyDeviceToHost);
    for (int m = 0; m <= l; m++)
        for (int Block=0; Block<k3gs; Block++)
            A_h[BaseAddress + m] = Complex_add(A_h[BaseAddress + m], PartialSum_h[Block*(l+1) + m]);
}

void etics::scf::CalculateCoefficients(Complex *A_h) {
    memset(A_h, 0, (NMAX+1)*(LMAX+1)*(LMAX+2)/2 * sizeof(Complex));
    for (int l = 0; l <= LMAX; l++) {
        if (l > 0) CalculatePhi0l<<<128,128>>>(l); // wouldn't it make sense just putting it after the n-loop finishes? Probably not becasue then we need to skip at the last iter
        for (int n = 0; n <= NMAX; n++) {
            CalculateCoefficients(n, l, A_h);
        }
    }
}

template<int Mode>
__device__ void etics::scf::CalculateGravityTemplate(int i, Complex *A, vec3 *F, Real *Potential) {
// it gets A as parameter because it can be either on host or device
#warning !!! This cannot really be a host function because it needs device cahce, angular coefficients which are on device!!
// 0 = both force and potential, 1 = only force, 2 = only pot
#define A(n,l,m) A[n*(LMAX+1)*(LMAX+2)/2 + l*(l+1)/2 + m]
    Real dPhiLeft;
    Real dPhiLeftMul;
    Real dPhiRight;
    Real dPhiRightAdd;
    Real dPhi;
    Real RadialPart2;
    Real PlmDerivTheta;

    Real Pot = 0;
    Real Fr = 0, Ftheta = 0, Fphi = 0;
    Real xi = Cache.xi[i];
    Real OneOverXiPlusOne = 1/(1+xi);
    Real r_I = (1-xi)*OneOverXiPlusOne;
    Real r = 1/r_I; // It's quite likely we can do everything without r.
    Real costheta = Cache.costheta[i];
    Real sintheta_I = rsqrt(1-costheta*costheta); // faster than using cachei // You sure??? in K3 it's the opposite

    Complex ExponentTmp[LMAX];
    Complex Exponent = Complex_conj(Cache.Exponent[i]);
    ExponentTmp[0] = Exponent;
    for (int m = 1; m < LMAX; m++) ExponentTmp[m] = Complex_mul(ExponentTmp[m-1],Exponent);
    if (Mode != 2) {
        Real xi2 = xi*xi;
        Real xi3 = xi2*xi;
        dPhiLeft = -0.25*OneOverXiPlusOne;
        dPhiLeftMul = 0.25*(1-xi2);
        dPhiRight = xi3 - xi2 - xi + 1;
        dPhiRightAdd = 2*(xi3 - 2*xi2 + xi);
    }
    Real Phi0l = 1/(1+r);
    Real tmp1 = Phi0l*Phi0l*r;
    for (int l = 0; l <= LMAX; l++) {
        if (Mode != 2) {
            if (l > 0) {
                dPhiLeft  *= dPhiLeftMul;
                dPhiRight += dPhiRightAdd;
            }
        }
        if (Mode != 2) dPhi = dPhiLeft * dPhiRight;
        for (int n = 0; n <= NMAX; n++) {
            Real Wnl, Wprev1, Wprev2;
            if (n == 0)      Wnl = 1;
            else if (n == 1) {Wnl = (4*l+3)*xi; Wprev2 = Wnl;}
            else if (n == 2) {Wnl = -(2*l+1.5)+( 8*l*(l+2) +7.5)*xi*xi; Wprev1 = Wnl;}
            else {
                Wnl = (xi*(2*n+4*l+1)*Wprev1 - (n+4*l+1)*Wprev2)/(Real)n;
                Wprev2 = Wprev1;
                Wprev1 = Wnl;
            }

            Real Wderiv = 0;
            if (n == 1) {Wderiv = 4*l + 3;}
            else if (n > 1) {
                Wderiv = (-n*xi*Wnl + (n+4*l+2)*Wprev2)/(1-xi*xi);
            } // From an unknown reason it's faster to have this Block separate from the previous one.

            Real RadialPart  = - SQRT_4_PI * Phi0l * Wnl;
            if (Mode != 2) RadialPart2 = SQRT_4_PI * (dPhi*Wnl + Phi0l*Wderiv*2/pow(1+r,2));

            Real Plm = Pl(l, costheta);
            Real tmp2 = Complex_real(A(n,l,0)) * AngCoeff[(l+1)*l/2] * Plm;
            if (Mode != 1) Pot += RadialPart  * tmp2;
            if (Mode != 2) Fr  += RadialPart2 * tmp2;

            if (l == 0) continue;

            //////////////////////////////// ugly fix
            if ((costheta < -0.999) || (costheta > +0.999)) {
                continue;
            }
            //////////////////////////////// ugly fix

            // The Block below is l>=1, m=0.
            if (Mode != 2) {
                PlmDerivTheta = (costheta*Plm - Pl(l-1, costheta))*l*sintheta_I; //TODO check if storing Pl(l-1) somewhere makes it faster
                Ftheta += - PlmDerivTheta * AngCoeff[(l+1)*l/2] * Complex_real(A(n,l,0)) * RadialPart * r_I;
            }

            // The Block below is l>=1, m=1.
            if (Mode == 2) PlmDerivTheta = (costheta*Plm - Pl(l-1, costheta))*l*sintheta_I; //TODO see above regarding storing Pl(l-1)
            Real Plm_prev1 = Plm;
            Plm = PlmDerivTheta; // PlmDerivTheta equals Plm for m=1.
            if (Mode != 2) PlmDerivTheta = - Plm*costheta*sintheta_I - l*(l+1)*Plm_prev1;
            tmp2 = 2 * AngCoeff[(l+1)*l/2+1];
            Complex tmp3 = Complex_mul(ExponentTmp[0], A(n,l,1));
            Complex tmp4 = make_Complex(tmp2 * tmp3.x, tmp2 * tmp3.y);
            Complex tmp5 = make_Complex(Plm *  tmp4.x, Plm *  tmp4.y);
            Complex tmp6 = make_Complex(RadialPart * tmp5.x, RadialPart * tmp5.y);
            if (Mode != 1) Pot += Complex_real(tmp6);
            if (Mode != 2) {
                Fr +=       RadialPart2 * Complex_real(tmp5);
                Fphi +=     Complex_imag(tmp6) * sintheta_I * r_I;
                Ftheta += - RadialPart * PlmDerivTheta *  Complex_real(tmp4) * r_I;
            }

            if (l == 1) continue;

            for (int m = 2; m <= l; m++) {
                Real Plm_prev2 = Plm_prev1;
                Plm_prev1 = Plm;
                Plm = - 2*(m-1)*costheta*sintheta_I*Plm_prev1 - (l+m-1)*(l-m+2)*Plm_prev2;
                tmp2 = 2 * AngCoeff[(l+1)*l/2+m];
                tmp3 = Complex_mul(ExponentTmp[m-1], A(n,l,m));
                tmp4 = make_Complex(tmp2 * tmp3.x, tmp2 * tmp3.y);
                tmp5 = make_Complex(Plm *  tmp4.x, Plm *  tmp4.y);
                tmp6 = make_Complex(RadialPart * tmp5.x, RadialPart * tmp5.y);
                if (Mode != 1) Pot  += Complex_real(tmp6);
                if (Mode != 2) {
                    PlmDerivTheta = - m*Plm*costheta*sintheta_I - (l+m)*(l-m+1)*Plm_prev1;
                    Fr +=       RadialPart2 * Complex_real(tmp5);
                    Fphi +=     m * Complex_imag(tmp6) * sintheta_I * r_I;
                    Ftheta += - RadialPart * PlmDerivTheta *  Complex_real(tmp4) * r_I;
                }
            }
        }
        Phi0l *= tmp1;
    }

    if (Mode != 2) {
        Real sintheta = 1/sintheta_I;
        Real tanphi = Exponent.y/Exponent.x;
        Real cosphi = ((Exponent.x >= 0)?(+1):(-1)) * rsqrt(1+tanphi*tanphi); // no simpler way to get sign bit?
        Real sinphi = tanphi*cosphi;
        *F = vec3(sintheta*cosphi*Fr + costheta*cosphi*Ftheta - sinphi*Fphi, sintheta*sinphi*Fr + costheta*sinphi*Ftheta + cosphi*Fphi,   costheta*Fr - sintheta*Ftheta);
    }
    if (Mode != 1) *Potential = Pot;
#undef A
}

__global__ void etics::scf::CalculateGravityFromCoefficients(Real *Potential, vec3 *F) { // formerly "Kernel4"
#define A(n,l,m) A[n*(LMAX+1)*(LMAX+2)/2 + l*(l+1)/2 + m]
#ifdef A_ON_SHARED_MEMORY
    __shared__ Complex Buffer[(NMAX+1)*(LMAX+1)*(LMAX+2)/2];
    if (threadIdx.x < warpSize) {
        for(int i = threadIdx.x; i  < (NMAX+1)*(LMAX+1)*(LMAX+2)/2; i += warpSize) {
            Buffer[i] = A[i];
        }
    }
    __syncthreads();
    #define A(n,l,m) Buffer[n*(LMAX+1)*(LMAX+2)/2 + l*(l+1)/2 + m]
#endif

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    while (i < Cache.N) {
        CalculateGravityTemplate<0>(i, A, &F[i], &Potential[i]);
#warning if we have A_ON_SHARED_MEMORY the above won't work
        i += blockDim.x * gridDim.x;
    }
}
#undef A

void etics::scf::SendCoeffsToGPU(Complex *A_h) {
        hipMemcpyToSymbol(HIP_SYMBOL(A), A_h, (NMAX+1)*(LMAX+1)*(LMAX+2)/2 * sizeof(Complex));
}

void etics::scf::CalculateGravity(Particle *P, int N, Real *Potential, vec3 *F) {
    LoadParticlesToCache<<<128,128>>>(P, N);
    CalculateCoefficients(A_h);
    Complex ATotal[(NMAX+1)*(LMAX+1)*(LMAX+2)/2];
    MPI_Allreduce(&A_h, &ATotal, (NMAX+1)*(LMAX+1)*(LMAX+2)/2*2, MPI_ETICS_REAL, MPI_SUM, MPI_COMM_WORLD);
    std::copy ( ATotal, ATotal+(NMAX+1)*(LMAX+1)*(LMAX+2)/2, A_h);
#warning not really need this copy, just calculate the coefficients in some local array, then sum it into a global array (A_h or somthing) and copy it to GPUs
//     hipMemcpyToSymbol(HIP_SYMBOL(A), A_h, (NMAX+1)*(LMAX+1)*(LMAX+2)/2 * sizeof(Complex));
    SendCoeffsToGPU(A_h);
    CalculateGravityFromCoefficients<<<k4gs,k4bs>>>(Potential, F);
}

namespace etics {
        namespace scf {
        int blockSizeToDynamicSMemSize(int BlockSize) { // Should be a lambda function
            return (LMAX+1)*sizeof(Complex)*BlockSize;
        }
    }
}

void etics::scf::Init(int N, int k3gs_new, int k3bs_new, int k4gs_new, int k4bs_new) {
    if ((k3gs_new==0) || (k3bs_new==0)) {
        cerr << "Warning: launch configuration for CalculateCoefficientsPartial(...) is unspecified; performance can be improved by optimizing it for this device." << endl;
        int blockSize;
        int minGridSize;
        int gridSize;
        hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, CalculateCoefficientsPartial, blockSizeToDynamicSMemSize, 128);
        cerr << "Warning: setting blockSizeLimit=128 for hipOccupancyMaxPotentialBlockSizeVariableSMem." << endl;
        gridSize = minGridSize;
        cerr << "Using the following launch configuration: <<<" << gridSize << "," << blockSize << ">>>" << endl;
        k3gs = gridSize;
        k3bs = blockSize;
    } else {
        k3gs = k3gs_new;
        k3bs = k3bs_new;
    }

    if ((k4gs_new==0) || (k4bs_new==0)) {
        cerr << "Warning: launch configuration for CalculateGravityFromCoefficients is unspecified; performance can be improved by optimizing it for this device." << endl;
        int blockSize;
        int minGridSize;
        int gridSize;
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, CalculateGravityFromCoefficients, 0, N);
        gridSize = (N + blockSize - 1) / blockSize;
        cerr << "Using the following launch configuration: <<<" << gridSize << "," << blockSize << ">>>" << endl;
        k4gs = gridSize;
        k4bs = blockSize;
    } else {
        k4gs = k4gs_new;
        k4bs = k4bs_new;
    }

    RadialCoefficients(RadCoeff_h);
    hipMemcpyToSymbol(HIP_SYMBOL(RadCoeff), RadCoeff_h, (NMAX+1)*(LMAX+1) * sizeof(Real));
    AngularCoefficients(AngCoeff_h);
    hipMemcpyToSymbol(HIP_SYMBOL(AngCoeff), AngCoeff_h, (LMAX+1)*(LMAX+2)/2 * sizeof(Real));
    InitializeCache(N);
    hipMemcpyToSymbol(HIP_SYMBOL(Cache), &Cache_h, sizeof(CacheStruct));
    PartialSum_h = (Complex*)malloc(k3gs*(LMAX+1)*sizeof(Complex)); // why not use "new"?
    hipMalloc((void**)&PartialSum, k3gs*(LMAX+1)*sizeof(Complex));
}
